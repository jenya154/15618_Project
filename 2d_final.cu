#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <stdio.h>
#include "rdtsc.h"
#include "CycleTimer.h"

// 3 x 3 convolutional mask
#define MASK_DIM 3

__global__ void convolution_2d(float* mask, float *matrix, float *result, int matrix_size, int result_size, int dimensions) {
 
      // Calculate the global thread positions in the result array
      int row = blockIdx.y * blockDim.y + threadIdx.y;
      int col = blockIdx.x * blockDim.x + threadIdx.x;

     // Temp value for accumulating the result
      int temp = 0;

    
    // Iterate over all the rows
    for (int i = 0; i < MASK_DIM; i++) {
        // Go over each column
        for (int j = 0; j < MASK_DIM; j++) {
            //matrix[(((i * sizeMatrix)+ j)*dimension) + d + kx + ky
            // Accumulate result
            temp += matrix[(row * matrix_size)+col+i+j] * mask[(i * MASK_DIM + j)];
        }
    }


     __syncthreads();
     // Write back the result
     result[(row * result_size + col)] = temp;
  
}

// Initializes an n x n input matrix with value of 1
void init_input(float *m, int n, int dimensions) {

    for(int i = 0; i < n * n * dimensions ; ++i) {
        m[i] = 1.0;     
}
}

// Initializes filter with value of 2
void init_mask(float *m, int n, int dimensions) {
    for (int i = 0; i < n * n * dimensions; ++i) {
        m[i] = 2.0;   
    }
  }

int main() {
  
  //Dimensions of the input matrix
  int N = 8;

  //Number of channels
  int dimensions = 8;

  long long sum1 = 0;
  tsc_counter t0, t1;

  // Size of the matrix (in bytes)
  size_t bytes_n = dimensions*N * N * sizeof(float);

  //Size of the result matrix
  size_t result_size = N - MASK_DIM + 1;

  //Total number of bytes in the result array
  size_t bytes_result = dimensions*result_size * result_size *sizeof(float);

  // Allocate the matrix and initialize it
  float *matrix = new float[dimensions*N * N];
  float *result = new float[dimensions*result_size * result_size];
  init_input(matrix, N, dimensions);

  // Size of the mask in bytes
  size_t bytes_m = dimensions*MASK_DIM * MASK_DIM * sizeof(float);

  // Allocate the mask and initialize it
  float *h_mask = new float[dimensions*MASK_DIM * MASK_DIM];
  init_mask(h_mask, MASK_DIM, dimensions);

  // Allocate device memory
  float *d_matrix;
  float *d_result;
  float* d_mask;
  hipMalloc(&d_matrix, bytes_n);
  hipMalloc(&d_result, bytes_result);
  hipMalloc(&d_mask, bytes_m);


  // Calculate grid dimensions
  int THREADS = result_size;
  int BLOCKS = dimensions;//(N + THREADS - 1) / THREADS;

  // Dimension launch arguments
  dim3 block_dim(THREADS, THREADS);
  dim3 grid_dim(BLOCKS,BLOCKS);

  RDTSC(t0);

  // Copy data to the device
  hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);
  hipMemcpy(d_mask, h_mask, bytes_m, hipMemcpyHostToDevice);

  // Perform 2D convolution
  convolution_2d<<<grid_dim, block_dim>>>(d_mask, d_matrix, d_result, N, result_size, dimensions);
  
  // Copy the result back to the CPU
  hipMemcpy(result, d_result,bytes_result , hipMemcpyDeviceToHost);

  RDTSC(t1);
  sum1 += (COUNTER_DIFF(t1, t0, CYCLES));

  printf("Average time: %lf cycles\n", ((double) (sum1 / ((double) runs))));

 //Printing output
for(int d = 0; d<dimensions; d++){
    for(int i = 0; i<result_size;i++){
        for(int j= 0; j<result_size;j++){
            printf("%f\t",result[ d*result_size*result_size + i*result_size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

  // Free the memory we allocated
  delete[] matrix;
  delete[] result;
  delete[] h_mask;

  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}
